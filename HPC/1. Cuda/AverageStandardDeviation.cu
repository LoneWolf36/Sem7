#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <numeric>
#include <time.h>
#include <math.h>
#include <stdio.h>
using namespace std;

__global__ void sum(double* input)
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			input[fst] += input[snd];
		}
		step_size <<= 1;
		number_of_threads >>= 1;
	}
}

__global__ void variance(int* n, double *x, double *mean)
{
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < *n; i+= stride) {
		x[i] = (x[i] - *mean) ;
		x[i] = x[i] * x[i];
	}
}

double calculateMean(int count , double *h) {

	const int size = count * sizeof(double);
	double* d;

	//mean
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	sum<<<1, count / 2 >>> (d);
	double result;
	hipMemcpy(&result, d, sizeof(double), hipMemcpyDeviceToHost);
	result /= count;
	// cout << "\nAverage is " << result << endl;
	hipFree(d);
	return result;
}

double* calculateVariance(int *count , double *h , double *mean) {

	const int size = (*count) * sizeof(double);
	double* d ;
	int* n;
	double *tempMean;

	hipMalloc(&d, size);
	hipMalloc(&n, sizeof(int));
	hipMalloc(&tempMean, sizeof(double));
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	hipMemcpy(n, count, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(tempMean, mean, sizeof(double), hipMemcpyHostToDevice);
	variance<<<1, (*count)/2 >>> (n , d  , tempMean);
	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
	hipFree(d);
	hipFree(n);
	hipFree(tempMean);
	return h;
}

int main()
{
	const int count = 8;      // size of array
	srand(2);
    double h[count];
    cout << "Elements of array are : " << endl;
	for ( int i = 0 ; i < count ; i ++ ) {
		h[i] = rand()%10;
		cout << h[i] << "\t" ;
    }
	int tempCount = count ;
	double mean = calculateMean(count , h);
	cout << "\nMean is : " << mean << endl;
	double *res = calculateVariance(&tempCount , h , &mean);
	// for ( int i = 0 ; i < count ; i ++ ) {
	//
    //         cout << res[i] << "\t" ;
	//
    // }
	double variance = calculateMean(count , res);
	cout << "Variance is : " << variance << endl;
	cout << "Standard Deviation is : " << sqrt(variance) << endl;
	delete[] &h;
	return 0;
}
