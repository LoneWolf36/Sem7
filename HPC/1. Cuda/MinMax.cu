#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <numeric>
#include <time.h>
#include <math.h>
using namespace std;

__global__ void max(double* input)
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			input[fst] = input[fst] >= input[snd] ? input[fst] : input[snd] ;
		}
		step_size <<= 1;
		number_of_threads >>= 1;
	}
}

__global__ void min(double* input)
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			input[fst] = input[fst] <= input[snd] ? input[fst] : input[snd] ;
		}
		step_size <<= 1;
		number_of_threads >>= 1;
	}
}


int main()
{
	const int count = 10;      // size of array
	srand(2);
    double h[count];
    for ( int i = 0 ; i < count ; i ++ ) {
        h[i] = rand();
        cout << h[i] << "\t" ;
    }

    const int size = count * sizeof(double);
	double* d;

	// Mean
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	max<<<1, count / 2 >>> (d);

	double result;
	hipMemcpy(&result, d, sizeof(double), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	cout << "\nMax is " << result << endl;
	hipFree(d);
    hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	min<<<1, count / 2 >>> (d);
	hipMemcpy(&result, d, sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cout << "Min is " << result << endl;
	hipFree(d);
/*
	// variance
	cudaMalloc(&d, size);
	cudaMemcpy(d, h, size, cudaMemcpyHostToDevice);

	double* temp ;
	cudaMalloc(&temp, sizeof(double));
	cudaMemcpy(temp, &res, sizeof(double), cudaMemcpyHostToDevice);

	var <<<1, count / 2 >>>(d, temp);

	cudaMemcpy(&res, d, sizeof(double), cudaMemcpyDeviceToHost);

	// res = res / count  ;

	cout << "Variance is " << res << endl;

	cudaFree(d);

	*/

	delete[] &h;
	return 0;
}
